/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "gtest/gtest.h"

#include "../src/overlapper_triggered.hpp"
#include "../src/overlapper_minimap.hpp"
#include "../src/chainer_utils.cuh"
#include <claraparabricks/genomeworks/cudamapper/overlapper.hpp>
#include <claraparabricks/genomeworks/cudamapper/utils.hpp>

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{

TEST(TestCudamapperOverlapperMinimap, scoring_beyond_bandwidth)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    CudaStream hip_stream           = make_cuda_stream();
    OverlapperMinimap overlapper(allocator, hip_stream.get());

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1000;

    Anchor anchor2;
    anchor2.query_read_id_           = 1;
    anchor2.target_read_id_          = 2;
    anchor2.query_position_in_read_  = 200;
    anchor2.target_position_in_read_ = 1100;

    Anchor anchor3;
    anchor3.query_read_id_           = 1;
    anchor3.target_read_id_          = 2;
    anchor3.query_position_in_read_  = 300;
    anchor3.target_position_in_read_ = 1200;

    Anchor anchor4;
    anchor4.query_read_id_           = 1;
    anchor4.target_read_id_          = 2;
    anchor4.query_position_in_read_  = 400;
    anchor4.target_position_in_read_ = 1300;
}

TEST(TestCudamapperOverlapperMinimap, scoring_beyond_max_dist)
{
}

TEST(TestCudamapperOverlapperMinimap, scoring_same_anchor)
{
}

TEST(TestCudamapperOverlapperMinimap, scoring_normal_case)
{
}

TEST(TestCudamapperOverlapperMinimap, Chaining)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    CudaStream hip_stream           = make_cuda_stream();
    OverlapperMinimap overlapper(allocator, hip_stream.get());
}

TEST(TestCumapperOverlapperMinimap, ChainingWithMM2Seeds)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    CudaStream hip_stream           = make_cuda_stream();

    // TODO: Add a permanent mm2 seed file. Should live in cudamapper/data
    std::string debug_file("../data/seed_debug_hg002_chr22_10521502.txt ");
    std::vector<chainerutils::seed_debug_entry> entries = chainerutils::read_minimap2_seed_chains(debug_file.c_str());

    std::shared_ptr<io::FastaParser> query_parser;
    std::shared_ptr<io::FastaParser> target_parser;
    query_parser  = io::create_kseq_fasta_parser("../data/test_hg002_chr22_10521502.fq", 15 + 5 - 1); // defaults taken from application parser
    target_parser = query_parser;                                                           // assume all to all


    for (auto const &entry : entries)
    {
        auto overlapper = std::make_unique<OverlapperMinimap>(allocator, hip_stream.get());
        std::vector<Overlap> overlaps;

        device_buffer<Anchor> d_anchors(entry.seeds.size(), allocator, hip_stream.get());
        
        cudautils::device_copy_n(entry.seeds.data(), entry.seeds.size(), d_anchors.data(), hip_stream.get());
        

        overlapper->get_overlaps(overlaps,
                                d_anchors,
                                false,
                                30,
                                30,
                                10,
                                0.9);
        
        Overlapper::post_process_overlaps(overlaps);
        ASSERT_EQ(overlaps.size(), entry.chains.size()); // Should this be a specific "set" of chains?
    }
    

}

} // namespace cudamapper
} // namespace genomeworks
} // namespace claraparabricks
